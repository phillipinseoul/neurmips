#include "utils.cuh"
#include <stdio.h>

void gpuAssert(hipError_t code)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr, "CUDA kernel failed : %s\n%s at L:%d in %s\n",
                hipGetErrorString(code), __PRETTY_FUNCTION__, __LINE__,
                __FILE__);                                              
        exit(-1);    
    }
}